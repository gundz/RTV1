#include <hip/hip_runtime.h>

void			do_cuda(t_data *data, int (*f)(t_data *, t_cuda *))
{
	static t_cuda	cuda = {NULL};
	const size_t	size = SDL_RY * SDL_RX * data->surf->format->BytesPerPixel;

	if (cuda.screen == NULL)
	{
		hipMalloc((void **)&cuda.screen, size);
		cuda.blocksize = dim3(32, 32);
		cuda.bx = (SDL_RX + cuda.blocksize.x - 1) / cuda.blocksize.x;
		cuda.by = (SDL_RY + cuda.blocksize.y - 1) / cuda.blocksize.y;
		cuda.gridsize = dim3(cuda.bx, cuda.by);
		cuda.rx = SDL_RX;
		cuda.ry = SDL_RY;
	}

	f(data, &cuda);
	hipDeviceSynchronize();

	SDL_LockSurface(data->surf);
	hipMemcpy(data->surf->pixels, cuda.screen, size, hipMemcpyDeviceToHost);
	SDL_UnlockSurface(data->surf);

	if (data->esdl->run == 0)
		hipFree(cuda.screen);
}